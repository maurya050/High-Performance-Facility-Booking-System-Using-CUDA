#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

__global__ void totalRequest(int max_fac, int N,int R, int *d_reqcen, int *d_reqfac, int *d_reqstart, int *d_reqslots, int* d_succesreqs, int* d_slots, int* d_success, int* d_failure)
{
    bool flag = true;
    int i, j, k = 0;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < max_fac)
    {
        k = 0;
        while(k < R ) 
        {
            int fac_num = d_reqcen[k] * 30 + d_reqfac[k];
            flag = true;
            int picked_slot = d_reqstart[k] + d_reqslots[k];
            if(fac_num == id)
            {
                //Checking, if the requested slot are in the range
                if(picked_slot <= 25)
                {
                    //i is 'requested slots' in the given facility (fac_num).
                    i = d_reqstart[k];
                    while( i < picked_slot )
                    {
                        int aval_cap = d_reqcen[k]*30*24 + d_reqfac[k]*24 + i-1;
                        //Checking , if the requested slot for the facility has the capacity greater than 0.
                        if(d_slots[aval_cap] >= 1)
                        {
                            atomicSub(&d_slots[aval_cap], 1);
                        }
                        else
                        {
                            //when the available capacity in the facility is less than 0.
                            j = d_reqstart[k];
                            flag = false;
                            while(j < i)
                            {
                                aval_cap = d_reqcen[k]*30*24 + d_reqfac[k]*24 + j-1;
                                atomicAdd(&d_slots[aval_cap],1);
                                j++;
                            }
                            break;
                        }
                        i++;  
                    }
                }
                else
                {
                    flag = false;
                }

    
                if (flag == true) 
                {
                    //center-wise, request success incremented.
                    atomicAdd(&d_succesreqs[d_reqcen[k]], 1);
                    //total request success incremented.
                    atomicAdd(&d_success[0], 1);
                }     
                else 
                {
                    //total request failure incremented.
                    atomicAdd(&d_failure[0], 1);
                }
            }
            k++;
        }  
    }
}


//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;


    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	int *offset = (int *) malloc ( (N) * sizeof (int) );
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N *  sizeof (int));  // stores capacities of each facility for every computer centre 

    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0, k3=0, *slots;
    slots=(int*)malloc(max_P * N * 24 * sizeof (int));
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      k3 = 0;
      offset[i]=k1;
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        for(int k4=0;k4<24;k4++){
            slots[centre[i]*30*24 + k3]=capacity[k2];
            k3++;
        }
        k2++; 
        
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }

	int *d_center, *d_facility, *d_facids , *d_capacity , *d_offset , *d_reqid, *d_reqcen, *d_reqfac, *d_reqstart, *d_reqslots, *d_slots, *d_totalreqs, *d_succesreqs,*d_success,*d_failure;
    hipMalloc(&d_succesreqs, (N) * sizeof(int));
    hipMalloc(&d_success,sizeof(int));
    hipMalloc(&d_totalreqs, (N) * sizeof(int));
    hipMalloc(&d_failure,  sizeof(int));
    hipMalloc(&d_succesreqs, (N) * sizeof(int));
	hipMalloc(&d_facids, (max_P*N) * sizeof(int));
	hipMalloc(&d_capacity, (max_P*N) * sizeof(int));
	hipMalloc(&d_offset, (N) * sizeof(int));
	hipMalloc(&d_reqid, (R) * sizeof(int));	
    hipMalloc(&d_reqcen, (R) * sizeof(int));
	hipMalloc(&d_reqfac, (R) * sizeof(int));
	hipMalloc(&d_reqstart, (R) * sizeof(int));
    hipMalloc(&d_reqslots, (R) * sizeof(int));
	hipMalloc(&d_slots, (max_P*N*24) * sizeof(int));
    hipMalloc(&d_center, (N) * sizeof(int));
	hipMalloc(&d_facility, (N) * sizeof(int));

    // Copy memory from host to device
	hipMemcpy(d_reqid, req_id, R * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_reqcen, req_cen, R * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_capacity, capacity, max_P * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_offset, offset,  N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_reqfac, req_fac, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_slots, slots, N*max_P*24 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_reqslots, req_slots, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_success, &success, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_failure, &fail,sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(d_reqstart, req_start, R * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_succesreqs, succ_reqs, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_center, centre, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_facility, facility, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_facids, fac_ids, max_P * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_totalreqs, tot_reqs, N * sizeof(int), hipMemcpyHostToDevice);

    //*********************************
    // Call the kernels here
    
    int max_fac = max_P * N;
    dim3 grid(ceil(float(N*max_P)/1024));
    dim3 block(1024);

    totalRequest<<<grid, block>>>(max_fac, N, R, d_reqcen, d_reqfac, d_reqstart, d_reqslots, d_succesreqs, d_slots, d_success, d_failure);
    hipMemcpy(&fail, d_failure, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&success, d_success, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();  
    
    //********************************

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");
    hipMemcpy(succ_reqs,d_succesreqs, N * sizeof(int), hipMemcpyDeviceToHost);
    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
      fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}